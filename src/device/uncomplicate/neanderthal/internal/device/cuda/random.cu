#include "hip/hip_runtime.h"
extern "C" {
    
#include "Random123/philox.h"
#include <stdint.h>
#include <float.h>
    
#ifndef M_2PI_FLOAT
#define M_2PI_FLOAT 6.2831855f
#endif

#ifndef M_2PI_DOUBLE
#define M_2PI_DOUBLE 6.283185307179586
#endif

    inline float u01_float(const uint32_t i) {
        return (0.5f + (i >> 9)) * FLT_EPSILON;
    }

    inline double u01_double(const uint64_t i) {
        return (0.5 + (i >> 12)) * DBL_EPSILON;
    }

    inline philox4x32_ctr_t rand_arr_32 (const uint64_t seed) {
        const uint32_t gidx = blockIdx.x * blockDim.x + threadIdx.x;
        const uint32_t gidy = blockIdx.y * blockDim.y + threadIdx.y;
        const uint32_t gidz = blockIdx.z * blockDim.z + threadIdx.z;
        philox4x32_key_t key;
        uint32_t* key_v = key.v;
        key_v[0] = seed;
        key_v[1] = 0xdecafaaa;
        philox4x32_ctr_t cnt;
        uint32_t* cnt_v = cnt.v;
        cnt_v[0] = gidx;
        cnt_v[1] = gidy;
        cnt_v[2] = gidz;
        cnt_v[3] = 0xbeeff00d;
        const philox4x32_ctr_t rand = philox4x32(cnt, key);
        return rand;
    }

    inline philox4x64_ctr_t rand_arr_64 (const uint64_t seed) {
        const uint64_t gidx = blockIdx.x * blockDim.x + threadIdx.x;
        const uint64_t gidy = blockIdx.y * blockDim.y + threadIdx.y;
        const uint64_t gidz = blockIdx.z * blockDim.z + threadIdx.z;
        philox4x64_key_t key;
        uint64_t* key_v = key.v;
        key_v[0] = seed;
        key_v[1] = 0xdecafaaa;
        philox4x64_ctr_t cnt;
        uint64_t* cnt_v = cnt.v;
        cnt_v[0] = gidx;
        cnt_v[1] = gidy;
        cnt_v[2] = gidz;
        cnt_v[3] = 0xbeeff00d;
        const philox4x64_ctr_t rand = philox4x64(cnt, key);
        return rand;
    }

    inline void box_muller_float(const uint32_t* i, float* g) {
        g[0] = sinf(M_2PI_FLOAT * u01_float(i[0]))
            * sqrtf(-2.0f * logf(u01_float(i[1])));
        g[1] = cosf(M_2PI_FLOAT * u01_float(i[0]))
            * sqrtf(-2.0f * logf(u01_float(i[1])));
        g[2] = sinf(M_2PI_FLOAT * u01_float(i[2]))
            * sqrtf(-2.0f * logf(u01_float(i[3])));
        g[3] = cosf(M_2PI_FLOAT * u01_float(i[2]))
            * sqrtf(-2.0f * logf(u01_float(i[3])));
    }

    inline void box_muller_double(const uint64_t* i, double* g) {
        g[0] = sin(M_2PI_DOUBLE * u01_double(i[0]))
            * sqrt(-2.0f * log(u01_double(i[1])));
        g[1] = cos(M_2PI_DOUBLE * u01_double(i[0]))
            * sqrt(-2.0f * log(u01_double(i[1])));
        g[2] = sin(M_2PI_DOUBLE * u01_double(i[2]))
            * sqrt(-2.0f * log(u01_double(i[3])));
        g[3] = cos(M_2PI_DOUBLE * u01_double(i[2]))
            * sqrt(-2.0f * log(u01_double(i[3])));
    }
    
    __global__ void vector_uniform_float (const int n, const uint64_t seed,
                                          const float lower, const float upper,
                                          float* x, const uint32_t offset_x, const uint32_t stride_x) {

        const uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;
        const int i = gid * 4;

        const philox4x32_ctr_t rand = rand_arr_32(seed);
        const float low = lower;
        const float upplow = upper - low;

        const int limit = (i + 3) < n ? 4 : n - i;
        for (int j = 0; j < limit; j++) {
            x[offset_x + ((i + j) * stride_x)] = u01_float(rand.v[j]) * upplow + low;
        }
    }

    __global__ void vector_uniform_double (const int n, const uint64_t seed,
                                           const double lower, const double upper,
                                           double* x, const uint32_t offset_x, const uint32_t stride_x) {

        const uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;
        const int i = gid * 4;

        const philox4x64_ctr_t rand = rand_arr_64(seed);
        const double low = lower;
        const double upplow = upper - low;

        const int limit = (i + 3) < n ? 4 : n - i;
        for (int j = 0; j < limit; j++) {
            x[offset_x + ((i + j) * stride_x)] = u01_double(rand.v[j]) * upplow + low;
        }
    }

    __global__ void vector_normal_float (const int n, const uint64_t seed,
                                   const float mu, const float sigma,
                                   float* x, const uint32_t offset_x, const uint32_t stride_x) {

        const uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;
        const int i = gid * 4;
        const philox4x32_ctr_t rand = rand_arr_32(seed);
        float g[4];
        box_muller_float(rand.v, g);
        const int limit = (i + 3) < n ? 4 : n - i;
        for (int j = 0; j < limit; j++) {
            x[offset_x + ((i + j) * stride_x)] = g[j] * sigma + mu;
        }
    }

    __global__ void vector_normal_double (const int n, const uint64_t seed,
                                          const double mu, const double sigma,
                                          double* x, const uint32_t offset_x, const uint32_t stride_x) {

        const uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;
        const int i = gid * 4;
        const philox4x64_ctr_t rand = rand_arr_64(seed);
        double g[4];
        box_muller_double(rand.v, g);
        const int limit = (i + 3) < n ? 4 : n - i;
        for (int j = 0; j < limit; j++) {
            x[offset_x + ((i + j) * stride_x)] = g[j] * sigma + mu;
        }
    }

}
