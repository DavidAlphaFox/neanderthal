
#include <hip/hip_runtime.h>
extern "C" {

#ifndef NUMBER
#define NUMBER float
#endif

    __global__ void vector_equals (const int n,
                                   const NUMBER* x, const int stride_x,
                                   const NUMBER* y, const int stride_y,
                                   int* eq_flag) {

        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            const int ix = gid * stride_x;
            const int iy = gid * stride_y;
            if (x[ix] != y[iy]) {
                eq_flag[0]++;
            }
        }
    }

    __global__ void vector_copy (const int n,
                                 const NUMBER* x, const int stride_x,
                                 NUMBER* y, const int stride_y) {

        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            const int ix = gid * stride_x;
            const int iy = gid * stride_y;
            y[iy] = x[ix];
        }
    }

    __global__ void vector_swap (const int n,
                                 NUMBER* x, const int stride_x,
                                 NUMBER* y, const int stride_y) {

        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            const int ix = gid * stride_x;
            const int iy = gid * stride_y;
            const NUMBER val = y[ix];
            y[iy] = x[ix];
            x[ix] = val;
        }
    }

    __global__ void vector_set (const int n, const NUMBER val, NUMBER* x, const int stride_x) {
        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            x[gid * stride_x] = val;
        }
    }

}
