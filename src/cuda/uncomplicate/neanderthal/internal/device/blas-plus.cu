#include "hip/hip_runtime.h"
extern "C" {
    
#ifndef REAL
#define REAL float
#endif

    __global__ void vector_equals (const int n,
                                   const REAL* x, const int offset_x, const int stride_x,
                                   const REAL* y, const int offset_y, const int stride_y,
                                   int* eq_flag) {

        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            const int ix = offset_x + gid * stride_x;
            const int iy = offset_y + gid * stride_y;
            if (x[ix] != y[iy]) {
                eq_flag[0]++;
            }
        }
    }


    __global__ void vector_copy (const int n,
                                 const REAL* x, const int offset_x, const int stride_x,
                                 REAL* y, int offset_y, int stride_y) {

        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            const int ix = offset_x + gid * stride_x;
            const int iy = offset_y + gid * stride_y;
            y[iy] = x[ix]; 
        }
    }

    
    __global__ void vector_sum (const int n,
                                const REAL* x, const int offset_x, const int stride_x,
                                ACCUMULATOR* acc) {

        int gid = (blockIdx.x * blockDim.x + threadIdx.x);
        ACCUMULATOR sum = block_reduction_sum( (gid < n) ? x[offset_x + gid * stride_x] : 0.0);
        if (threadIdx.x == 0) {
            acc[blockIdx.x] = sum;
        }
        
    }
    
    __global__ void vector_set (const int n, const REAL val,
                                REAL* x, const int offset_x, const int stride_x) {
        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            x[offset_x + gid * stride_x] = val;
        }
    }

    __global__ void vector_axpby (const int n,
                                  const REAL alpha, const REAL* x, const int offset_x, const int stride_x,
                                  const REAL beta, REAL* y, int offset_y, int stride_y) {

        const int gid = blockIdx.x * blockDim.x + threadIdx.x;
        if (gid < n) {
            const int ix = offset_x + gid * stride_x;
            const int iy = offset_y + gid * stride_y;
            y[iy] = alpha * x[ix] + beta * y [iy]; 
        }
    }

    __global__ void ge_equals_no_transp (const int sd, const int fd,
                                         const REAL* a, const int offset_a, const int ld_a,
                                         const REAL* b, const int offset_b, const int ld_b,
                                         int* eq_flag) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < fd);
        if (valid) {
            const int ia = offset_a + gid_0 + gid_1 * ld_a;
            const int ib = offset_b + gid_0 + gid_1 * ld_b;
            if (a[ia] != b[ib]){
                eq_flag[0]++;
            }
        }
    }

    __global__ void ge_equals_transp (const int sd, const int fd,
                                      const REAL* a, const int offset_a, const int ld_a,
                                      const REAL* b, const int offset_b, const int ld_b,
                                      int* eq_flag) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < fd);
        if (valid) {
            const int ia = offset_a + gid_0 + gid_1 * ld_a;
            const int ib = offset_b + gid_1 + gid_0 * ld_b;
            if (a[ia] != b[ib]){
                eq_flag[0]++;
            }
        }
    }

    __global__ void ge_swap_no_transp (const int sd, const int fd,
                                       REAL* a, const int offset_a, const int ld_a,
                                       REAL* b, const int offset_b, const int ld_b) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < fd);
        if (valid) {
            const int ia = offset_a + gid_0 + gid_1 * ld_a;
            const int ib = offset_b + gid_0 + gid_1 * ld_b;
            const REAL c = b[ib];
            b[ib] = a[ia];
            a[ia] = c;
        }
    }

    __global__ void ge_swap_transp (const int sd, const int fd,
                                    REAL* a, const int offset_a, const int ld_a,
                                    REAL* b, const int offset_b, const int ld_b) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < fd);
        if (valid) {
            const int ia = offset_a + gid_0 + gid_1 * ld_a;
            const int ib = offset_b + gid_1 + gid_0 * ld_b;
            const REAL c = b[ib];
            b[ib] = a[ia];
            a[ia] = c;
        }
    }

    __global__ void ge_set (const int sd, const int fd,
                            const REAL val, REAL* a, const int offset_a, const int ld_a) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < fd);
        if (valid) {
            a[offset_a + gid_0 + gid_1 * ld_a] = val;
        }
    }

    __global__ void uplo_equals_no_transp (const int sd, const int unit, const int bottom,
                                           const REAL* a, const int offset_a, const int ld_a,
                                           const REAL* b, const int offset_b, const int ld_b,
                                           int* eq_flag) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < sd);
        const bool check = valid &&
            ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);
        if (check) {
            const int ia = offset_a + gid_0 + gid_1 * ld_a;
            const int ib = offset_b + gid_0 + gid_1 * ld_b;
            if (a[ia] != b[ib]){
                eq_flag[0]++;
            }
        }
    }

    __global__ void uplo_equals_transp (const int sd, const int unit, const int bottom,
                                        const REAL* a, const int offset_a, const int ld_a,
                                        const REAL* b, const int offset_b, const int ld_b,
                                        int* eq_flag) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < sd);
        const bool check = valid &&
            ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);
        if (check) {
            const int ia = offset_a + gid_0 + gid_1 * ld_a;
            const int ib = offset_b + gid_1 + gid_0 * ld_b;
            if (a[ia] != b[ib]){
                eq_flag[0]++;
            }
        }
    }

    __global__ void uplo_copy_no_transp (const int sd, const int unit, const int bottom,
                                         const REAL* a, const int offset_a, const int ld_a,
                                         REAL* b, const int offset_b, const int ld_b) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < sd);
        const bool check = valid &&
            ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);
        if (check) {
            const int ia = offset_a + gid_0 + gid_1 * ld_a;
            const int ib = offset_b + gid_0 + gid_1 * ld_b;
            b[ib] = a[ia];
        }
    }

    __global__ void uplo_copy_transp (const int sd, const int unit, const int bottom,
                                      const REAL* a, const int offset_a, const int ld_a,
                                      REAL* b, const int offset_b, const int ld_b) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < sd);
        const bool check = valid &&
            ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);
        if (check) {
            const int ia = offset_a + gid_0 + gid_1 * ld_a;
            const int ib = offset_b + gid_1 + gid_0 * ld_b;
            b[ib] = a[ia];
        }
    }

    __global__ void uplo_swap_no_transp (const int sd, const int unit, const int bottom,
                                         REAL* a, const int offset_a, const int ld_a,
                                         REAL* b, const int offset_b, const int ld_b) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < sd);
        const bool check = valid &&
            ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);
        if (check) {
            const int ia = offset_a + gid_0 + gid_1 * ld_a;
            const int ib = offset_b + gid_0 + gid_1 * ld_b;
            const REAL c = b[ib];
            b[ib] = a[ia];
            a[ia] = c;
        }
    }

    __global__ void uplo_swap_transp (const int sd, const int unit, const int bottom,
                                      REAL* a, const int offset_a, const int ld_a,
                                      REAL* b, const int offset_b, const int ld_b) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < sd);
        const bool check = valid &&
            ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);
        if (check) {
            const int ia = offset_a + gid_0 + gid_1 * ld_a;
            const int ib = offset_b + gid_1 + gid_0 * ld_b;
            const REAL c = b[ib];
            b[ib] = a[ia];
            a[ia] = c;
        }
    }

    __global__ void uplo_axpby_no_transp (const int sd, const int unit, const int bottom,
                                          const REAL alpha, const REAL* a, const int offset_a, const int ld_a,
                                          const REAL beta, REAL* b, const int offset_b, const int ld_b) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < sd);
        const bool check = valid &&
            ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);
        if (check) {
            const int ia = offset_a + gid_0 + gid_1 * ld_a;
            const int ib = offset_b + gid_0 + gid_1 * ld_b;
            b[ib] = alpha * a[ia] + beta * b[ib];
        }
    }

    __global__ void uplo_axpby_transp (const int sd, const int unit, const int bottom,
                                       const REAL alpha, const REAL* a, const int offset_a, const int ld_a,
                                       const REAL beta, REAL* b, const int offset_b, const int ld_b) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < sd);
        const bool check = valid && (unit == 132)
            ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1;
        if (check) {
            const int ia = offset_a + gid_0 + gid_1 * ld_a;
            const int ib = offset_b + gid_1 + gid_0 * ld_b;
            b[ib] = alpha * a[ia] + beta * b[ib];
        }
    }

    __global__ void uplo_scal (const int sd, const int unit, const int bottom,
                               const REAL alpha, REAL* a, const int offset_a, const int ld_a) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < sd);
        const bool check = valid &&
            ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);
        if (check) {
            a[offset_a + gid_0 + gid_1 * ld_a] *= alpha;
        }
    }

    __global__ void uplo_set (const int sd, const int unit, const int bottom,
                              const REAL alpha, REAL* a, const int offset_a, const int ld_a) {
        const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
        const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
        const bool valid = (gid_0 < sd) && (gid_1 < sd);
        const bool check = valid &&
            ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);
        if (check) {
            a[offset_a + gid_0 + gid_1 * ld_a] = alpha;
        }
    }
    
}
